// https://www.jianshu.com/p/a0184e73a460

#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>

#define w 2000

struct Matrix
{
    int width;
    int height;
    float *elements;
};

__device__ float getElement(Matrix *A, int row, int col)
{
    return A->elements[row * A->width + col];
}

__device__ void setElement(Matrix *A, int row, int col, float value)
{
    A->elements[row * A->width + col] = value;
}

__global__ void matMulKernel(Matrix *A, Matrix *B, Matrix *C)
{
    float Cvalue = 0.0;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int col = threadIdx.x + blockIdx.x * blockDim.x;
	//int tid = threadIdx.x;
    for (int i = 0; i < A->width; ++i)
    {
        int index = (int)getElement(B, i, col) % 23;
        Cvalue += getElement(A, row, i) * getElement(B, i, col);
        // Cvalue += Cvalue / (double)3.1;
        //sudo /usr/local/cuda-10.2/bin/nvprof --metrics branch_efficiency  ./matrixMul
        //if (index == 17)
        //{
        //    for (int i = 0; i < 23; i++)
        //    {
        //        //Cvalue += Cvalue / (double)3.1;
        //    }
        //}
        Cvalue += Cvalue / (double)0.3456789;
        //Cvalue += Cvalue / (float)3.123234;
        //else
        //{
            //Cvalue += Cvalue / (double)3.2;
        //}
        //Cvalue += Cvalue / (double)3.22;
        //Cvalue += Cvalue / (double)3.23;
        //Cvalue += Cvalue / (float)3.1;
        

        //printf("%f \n", rsqrtf(4));
        //Cvalue += sqrtf(Cvalue);
        //Cvalue += acosf(Cvalue);
        //if (index == 0)
        {
            //Cvalue += sqrtf(Cvalue);
            //Cvalue += Cvalue / (double)3.1;
        }
        /*
        else if (index == 1)
        {
            Cvalue += Cvalue / (double)3.1415926;
        }
        else if (index == 2)
        {
            Cvalue += Cvalue / (double)2.71828;
        }
        else if (index == 3)
        {
            Cvalue += Cvalue / (double)1.4141;
        }
        else if (index == 4)
        {
            Cvalue += Cvalue / (double)1.4142;
        }
        else if (index == 5)
        {
            Cvalue += Cvalue / (double)1.4143;
        }
        else if (index == 6)
        {
            Cvalue += Cvalue / (double)1.4144;
        }
        else if (index == 7)
        {
            Cvalue += Cvalue / (double)1.4145;
        }
        else if (index == 8)
        {
            Cvalue += Cvalue / (double)1.4146;
        }
        else if (index == 9)
        {
            Cvalue += Cvalue / (double)1.4147;
        }
        else if (index == 10)
        {
            Cvalue += Cvalue / (double)1.4148;
        }
        else if (index == 11)
        {
            Cvalue += Cvalue / (double)1.4149;
        }
        else if (index == 12)
        {
            Cvalue += Cvalue / (double)1.41401;
        }
        else if (index == 13)
        {
            Cvalue += Cvalue / (double)1.4121;
        }
        else if (index == 14)
        {
            Cvalue += Cvalue / (double)1.4124;
        }
        else if (index == 15)
        {
            Cvalue += Cvalue / (double)1.41214;
        }
        else if (index == 16)
        {
            Cvalue += Cvalue / (double)3.14159246;
        }
        else if (index == 17)
        {
            Cvalue += Cvalue / (double)2.7182843;
        }
        else if (index == 18)
        {
            Cvalue += Cvalue / (double)1.414145;
        }
        else if (index == 19)
        {
            Cvalue += Cvalue / (double)1.414122;
        }
        else if (index == 20)
        {
            Cvalue += Cvalue / (double)1.41423;
        }
        else if (index == 21)
        {
            Cvalue += Cvalue / (double)1.41444;
        }
        else if (index == 22)
        {
            Cvalue += Cvalue / (double)1.41453;
        }
        else if (index == 23)
        {
            Cvalue += Cvalue / (double)1.43146;
        }
        else if (index == 24)
        {
            Cvalue += Cvalue / (double)1.24147;
        }
        else if (index == 25)
        {
            Cvalue += Cvalue / (double)1.14148;
        }
        else if (index == 26)
        {
            Cvalue += Cvalue / (double)1.41149;
        }
        else if (index == 27)
        {
            Cvalue += Cvalue / (double)1.414201;
        }
        else if (index == 28)
        {
            Cvalue += Cvalue / (double)(1.41218 + 28);
        }
        else if (index == 29)
        {
            Cvalue += Cvalue / (double)(1.41214 + 29);
        }
        else if (index == 30)
        {
            Cvalue += Cvalue / (double)(1.41214 + 30);
        }
        else if (index == 31)
        {
            Cvalue += Cvalue / (double)(1.41214 + 31);
        }*/
    }
    setElement(C, row, col, Cvalue);
}



void GPU_Test()
{
	int width = w;
    int height = w;
    Matrix *A, *B, *C;
    hipMallocManaged((void**)&A, sizeof(Matrix));
    hipMallocManaged((void**)&B, sizeof(Matrix));
    hipMallocManaged((void**)&C, sizeof(Matrix));

    int nBytes = width * height * sizeof(float);

    hipMallocManaged((void**)&A->elements, nBytes);
    hipMallocManaged((void**)&B->elements, nBytes);
    hipMallocManaged((void**)&C->elements, nBytes);

    A->height = height;
    A->width = width;
    B->height = height;
    B->width = width;
    C->height = height;
    C->width = width;

    for (int i = 0; i < width * height; ++i)
    {
        A->elements[i] = rand();// + 1.0;
        //printf("%d \n",((int)A->elements[i] % 23));
		B->elements[i] = rand();// + 2.0;
    }

    dim3 blockSize(32, 32);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    struct timeval t1,t2;
    gettimeofday(&t1,NULL);
    double timeuse;

    matMulKernel<<<gridSize, blockSize>>>(A, B, C);

    hipDeviceSynchronize();

    gettimeofday(&t2,NULL);
    timeuse = t2.tv_sec - t1.tv_sec + (t2.tv_usec - t1.tv_usec)/1000000.0;
    printf("GPU Use Time:%fs\n", timeuse);

}

int main()
{
	//CPU_Test();
	GPU_Test();
	return 0;
}
